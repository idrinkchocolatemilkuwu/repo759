#include "hip/hip_runtime.h"
#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

int main(int argc, char **argv){
	unsigned int n = atol(argv[1]);
	unsigned int R = atol(argv[2]);
	unsigned int threads_per_block = atol(argv[3]);

	//set up random number generators
	std::random_device entropy_source;
	std::mt19937_64 generator(entropy_source());
	const float min = -1.0, max = 1.0;
	std::uniform_real_distribution<float> dist_image(min,max);
	std::uniform_real_distribution<float> dist_mask(min,max);

	//initialize the arrays
	float *image, *output, *mask;	
	hipMallocManaged((void **)&image, n * sizeof(float));
	hipMallocManaged((void **)&output, n * sizeof(float));
	hipMallocManaged((void **)&mask, (2 * R + 1) * sizeof(float));
	for (unsigned int i = 0; i < n; i++){
		image[i] = dist_image(generator);
	}
	for (unsigned int i = 0; i < 2 * R + 1; i++){
		mask[i] = dist_mask(generator);
	}

	//set up cuda events to time
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//launch kernel and time the events
	hipEventRecord(start);
	stencil(image, mask, output, n, R, threads_per_block);
	hipEventRecord(stop);

	//calculate the elapsed time in ms
	float ms;
	hipEventElapsedTime(&ms, start, stop);

	//print out the last element of the output array and elapsed time
	printf("%f\n%f\n", output[n-1], ms);

	hipFree(image);
	hipFree(output);
	hipFree(mask);
	return 0;
}
