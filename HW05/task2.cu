#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

int main(int argc, char **argv){
	unsigned int n = atol(argv[1]);
	unsigned int block_dim = atol(argv[2]);

	//matmul_1
	int *A_in, *B_in, *C_in;
	hipMallocManaged((void **)&A_in, n * n * sizeof(int));
	hipMallocManaged((void **)&B_in, n * n * sizeof(int));
	hipMallocManaged((void **)&C_in, n * n * sizeof(int));
	//A_in will store 0,1,...,n * n - 1
	//B_in will be identity matrix
	for (unsigned int i = 0; i < n; ++i){
		for (unsigned int j = 0; j < n; ++j){
			A_in[i * n + j] = int(i * n + j);
			if (i == j){
				B_in[i * n + j] = 1;
			}else{
				B_in[i * n + j] = 0;
			}
		}
	}
	//set up cuda events to time
	hipEvent_t start_1;
	hipEvent_t stop_1;
	hipEventCreate(&start_1);
	hipEventCreate(&stop_1);
	//call matmul_1 and time the function
	hipEventRecord(start_1);
	matmul_1(A_in,B_in,C_in,n,block_dim);
	hipEventRecord(stop_1);
	hipEventSynchronize(stop_1);
	//calculate the elasped time in ms
	float ms_1;
	hipEventElapsedTime(&ms_1, start_1, stop_1);
	//print out the first element, last element and elapsed time
	printf("%i\n%i\n%f\n",C_in[0],C_in[n * n - 1],ms_1);
	//deallocate
	hipEventDestroy(start_1);
	hipEventDestroy(stop_1);
	hipFree(A_in);
	hipFree(B_in);
	hipFree(C_in);

	
	//matmul_2
	float *A_fl, *B_fl, *C_fl;
	hipMallocManaged((void **)&A_fl, n * n * sizeof(float));
	hipMallocManaged((void **)&B_fl, n * n * sizeof(float));
	hipMallocManaged((void **)&C_fl, n * n * sizeof(float));
	//A_fl will store 0,1,...,n * n - 1
	//B_fl will be identity matrix
	for (unsigned int i = 0; i < n; ++i){
		for (unsigned int j = 0; j < n; ++j){
			A_fl[i * n + j] = float(i * n + j);
			if (i == j){
				B_fl[i * n + j] = 1;
			}else{
				B_fl[i * n + j] = 0;
			}
		}
	}
	//set up cuda events to time
	hipEvent_t start_2;
	hipEvent_t stop_2;
	hipEventCreate(&start_2);
	hipEventCreate(&stop_2);
	//call matmul_2 and time the function
	hipEventRecord(start_2);
	matmul_2(A_fl,B_fl,C_fl,n,block_dim);
	hipEventRecord(stop_2);
	hipEventSynchronize(stop_2);
	//calculate the elapsed time in ms
	float ms_2;
	hipEventElapsedTime(&ms_2, start_2, stop_2);
	//print out the first element, last element and elapsed time
	printf("%f\n%f\n%f\n",C_fl[0],C_fl[n * n - 1],ms_2);
	//deallocate
	hipEventDestroy(start_2);
	hipEventDestroy(stop_2);
	hipFree(A_fl);
	hipFree(B_fl);
	hipFree(C_fl);


	//matmul_3
	double *A_db, *B_db, *C_db;
	hipMallocManaged((void **)&A_db, n * n * sizeof(double));
	hipMallocManaged((void **)&B_db, n * n * sizeof(double));
	hipMallocManaged((void **)&C_db, n * n * sizeof(double));
	//A_db will store 0,1,...,n * n - 1
	//B_db will be identity matrix
	for (unsigned int i = 0; i < n; ++i){
		for (unsigned int j = 0; j < n; ++j){
			A_db[i * n + j] = double(i * n + j);
			if (i == j){
				B_db[i * n + j] = 1;
			}else{
				B_db[i * n + j] = 0;
			}
		}
	}
	//set up cuda events to time
	hipEvent_t start_3;
	hipEvent_t stop_3;
	hipEventCreate(&start_3);
	hipEventCreate(&stop_3);
	//call matmul_3 and time the function
	hipEventRecord(start_3);
	matmul_3(A_db,B_db,C_db,n,block_dim);
	hipEventRecord(stop_3);
	hipEventSynchronize(stop_3);
	//calculate the elapsed time in ms
	float ms_3;
	hipEventElapsedTime(&ms_3, start_3, stop_3);
	//print out the first element, last element and elapsed time
	printf("%f\n%f\n%f",C_db[0],C_db[n * n - 1],ms_3);
	//deallocate
	hipEventDestroy(start_3);
	hipEventDestroy(stop_3);
	hipFree(A_db);
	hipFree(B_db);
	hipFree(C_db);

	return 0;
}

