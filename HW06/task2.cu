#include "hip/hip_runtime.h"
#include "scan.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

int main(int argc, char** argv) {

	int n = atol(argv[1]);
	int threads_per_block = atol(argv[2]);

	//set up random number generators
	std::random_device entropy_source;
	std::mt19937_64 generator(entropy_source());
	const float min = -1.0, max = 1.0;
	std::uniform_real_distribution<float> dist(min, max);

	//create input output
	//fill with random numbers
	float* input, *output;
	hipMallocManaged((void**)&input, n * sizeof(float));
	hipMallocManaged((void**)&output, n * sizeof(float));
	for (int i = 0; i < n; i++) {
		input[i] = dist(generator);
	}

	//set up cuda events to time
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//call the kernel and time it
	hipEventRecord(start);
	scan(input, output, n, threads_per_block);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	//calculate the elapsed time in ms
	float ms;
	hipEventElapsedTime(&ms, start, stop);

	//print out the last element and the time
	printf("%f\n%f", output[n - 1], ms);

	hipFree(input); hipFree(output);
	return 0;
}