#include "hip/hip_runtime.h"
#include "mmul.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <hipblas.h>

int main(int argc, char** argv) {

	int n = atol(argv[1]);
	int n_tests = atol(argv[2]);

	//set up random number generators
	std::random_device entropy_source;
	std::mt19937_64 generator(entropy_source());
	const float min = -1.0, max = 1.0;
	std::uniform_real_distribution<float> dist_a(min, max);
	std::uniform_real_distribution<float> dist_b(min, max);
	std::uniform_real_distribution<float> dist_c(min, max);

	//create A, B and C
	//fill with random numbers
	float* A, * B, * C;
	hipMallocManaged((void**)&A, n * n * sizeof(float));
	hipMallocManaged((void**)&B, n * n * sizeof(float));
	hipMallocManaged((void**)&C, n * n * sizeof(float));
	for (int i = 0; i < n * n; i++) {
		A[i] = dist_a(generator);
		B[i] = dist_b(generator);
		C[i] = dist_c(generator);
	}

	//set up cublas
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//set up cuda events to time
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//call mmul n_tests times and time it
	hipEventRecord(start);
	for (int i = 0; i < n_tests; i++) {
		mmul(handle, A, B, C, n);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	//calculate the elapsed time in ms
	float ms;
	hipEventElapsedTime(&ms, start, stop);

	//print out the average time taken by a single call to mmul
	float average_ms = ms / n_tests;
	printf("%f", average_ms);

	hipFree(A); hipFree(B); hipFree(C);
	hipblasDestroy(handle);
	return 0;
}